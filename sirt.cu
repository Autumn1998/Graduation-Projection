#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

//texture<double,2,hipReadModeElementType> tex_x;
//texture<double,2,hipReadModeElementType> tex_y;

__global__ void backProjOnGPU(Projection *prj,Volume *vol,double *x_coef,double *y_coef,float *prj_real,float *vol_real,float iter_step_length)
{
	double divisor;//分子
	double dividend;//分母
	int x = threadIdx.x+blockIdx.x*blockDim.x +vol->Xstart;
	int y = threadIdx.y+blockIdx.y*blockDim.y +vol->Ystart;
	int z = threadIdx.z+blockIdx.z*blockDim.z +vol->Zstart;
	//printf("%d %d\n ",y,z);
	if(x>=vol->Xend || y>=vol->Yend ||z>=vol->Zend) return;
	divisor = 0;
	dividend = 0;
	int index,angle,n;

	for(angle=0;angle<prj->AngN;angle++)
	{
		double res_x,res_y,x_min_del,y_min_del;
		int id = 4*angle,x_min,y_min;	
		res_x = x_coef[id]+x_coef[id+1]*x+x_coef[id+2]*y+x_coef[id+3]*z;
		res_y = y_coef[id]+y_coef[id+1]*x+y_coef[id+2]*y+y_coef[id+3]*z;	
		x_min = floor(res_x);
		y_min = floor(res_y);
		x_min_del = res_x - x_min;
		y_min_del = res_y - y_min;
	
		if(x_min>=0 && x_min<prj->X && y_min>=0 && y_min<prj->Y)//(x_min,y_min)
		{
			n = x_min + y_min*prj->X + angle*prj->X*prj->Y;
			divisor += (1-x_min_del)*(1-y_min_del)*prj_real[n];
			dividend += (1-x_min_del)*(1-y_min_del);
		}
		if(x_min+1>=0 && x_min+1<prj->X && y_min>=0 && y_min<prj->Y)//(x_min+1,y_min)
		{
			n = (x_min+1) + y_min*prj->X + angle*prj->X*prj->Y;
			divisor += x_min_del*(1-y_min_del)*prj_real[n];
			dividend += x_min_del*(1-y_min_del);
		}
		if(x_min>=0 && x_min<prj->X && y_min+1>=0 && y_min+1<prj->Y)//(x_min,y_min+1)
		{
			n = x_min + (y_min+1)*prj->X + angle*prj->X*prj->Y;
			divisor += (1-x_min_del)*y_min_del*prj_real[n];
			dividend += (1-x_min_del)*y_min_del;
		}
		if(x_min+1>=0 && x_min+1<prj->X && y_min+1>=0 && y_min+1<prj->Y)//(x_min+1,y_min+1)
		{
			n = (x_min+1)+ (y_min+1)*prj->X + angle*prj->X*prj->Y;
			divisor += x_min_del*y_min_del*prj_real[n];
			dividend += x_min_del*y_min_del;
		}
	}
	if(dividend!=0.0f)
	{
		index = (x-vol->Xstart)+(y-vol->Ystart)*vol->X+(z-vol->Zstart)*vol->X*vol->Y;
		vol_real[index] += (float)(divisor/dividend)*iter_step_length;
	}
}

__global__ void reProjOnGPU(Projection *prj,Volume *vol,double *x_coef,double *y_coef,float *vol_real,float *iter_prj_divisor,float *iter_prj_dividend)
{
	
	int x = threadIdx.x+blockIdx.x*blockDim.x +vol->Xstart;
	int y = threadIdx.y+blockIdx.y*blockDim.y +vol->Ystart;
	int z = threadIdx.z+blockIdx.z*blockDim.z +vol->Zstart;
	//printf("%d %d\n ",y,z);
	if(x>=vol->Xend || y>=vol->Yend ||z>=vol->Zend) return;	
	int index,angle,n;
	index = (x-vol->Xstart)+(y-vol->Ystart)*vol->X+(z-vol->Zstart)*vol->X*vol->Y;
	
	for(angle=0;angle<prj->AngN;angle++)
	{
		double res_x,res_y,x_min_del,y_min_del;
		int id = 4*angle,x_min,y_min;	
		res_x = x_coef[id]+x_coef[id+1]*x+x_coef[id+2]*y+x_coef[id+3]*z;
		res_y = y_coef[id]+y_coef[id+1]*x+y_coef[id+2]*y+y_coef[id+3]*z;	
		x_min = floor(res_x);
		y_min = floor(res_y);
		x_min_del = res_x - x_min;
		y_min_del = res_y - y_min;
		
		if(x_min>=0 && x_min<prj->X && y_min>=0 && y_min<prj->Y)//(x_min,y_min)
		{
			n = x_min + y_min*prj->X + angle*prj->X*prj->Y;
			atomicAdd(&iter_prj_divisor[n], (1-x_min_del)*(1-y_min_del)*vol_real[index]);
			atomicAdd(&iter_prj_dividend[n], (1-x_min_del)*(1-y_min_del));
		}
		if(x_min+1>=0 && x_min+1<prj->X && y_min>=0 && y_min<prj->Y)//(x_min+1,y_min)
		{
			n = (x_min+1) + y_min*prj->X + angle*prj->X*prj->Y;
			atomicAdd(&iter_prj_divisor[n], x_min_del*(1-y_min_del)*vol_real[index]);
			atomicAdd(&iter_prj_dividend[n], x_min_del*(1-y_min_del));
		}
		if(x_min>=0 && x_min<prj->X && y_min+1>=0 && y_min+1<prj->Y)//(x_min,y_min+1)
		{
			n = x_min + (y_min+1)*prj->X + angle*prj->X*prj->Y;
			atomicAdd(&iter_prj_divisor[n], (1-x_min_del)*y_min_del*vol_real[index]);
			atomicAdd(&iter_prj_dividend[n], (1-x_min_del)*y_min_del);
		}
		if(x_min+1>=0 && x_min+1<prj->X && y_min+1>=0 && y_min+1<prj->Y)//(x_min+1,y_min+1)
		{
			n = (x_min+1)+ (y_min+1)*prj->X + angle*prj->X*prj->Y;
			atomicAdd(&iter_prj_divisor[n], x_min_del*y_min_del*vol_real[index]);
			atomicAdd(&iter_prj_dividend[n], x_min_del*y_min_del);
		}
	}
}


__global__ void computePrjError(Projection *prj,float *prj_real,float *iter_prj_divisor,float *iter_prj_dividend)
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	if(x>=prj->X || y>=prj->Y ||z>=prj->AngN) return;
	int index;	
	index = x+y*prj->X+z*prj->X*prj->Y;
	if(iter_prj_dividend[index]!=0)
		iter_prj_divisor[index] /= iter_prj_dividend[index];
	iter_prj_divisor[index] = prj_real[index]-iter_prj_divisor[index];
}

__global__ void copyDataOnGPU(float *tar,float *sou,Volume *vol)
{
	int x = threadIdx.x+blockIdx.x*blockDim.x+vol->Xstart;
	int y = threadIdx.y+blockIdx.y*blockDim.y+vol->Ystart;
	int z = threadIdx.z+blockIdx.z*blockDim.z+vol->Zstart;
	//printf("%d %d\n ",y,z);
	if(x>=vol->Xend || y>=vol->Yend ||z>=vol->Zend) return;	
	int index = (x-vol->Xstart)+(y-vol->Ystart)*vol->X+(z-vol->Zstart)*vol->X*vol->Y;
	tar[index] = sou[index];
}


__global__ void maceOnGPU(float *v_data,float *w_data,float *z_data,Volume *vol,float GAMMA)
{
	int x = threadIdx.x+blockIdx.x*blockDim.x +vol->Xstart;
	int y = threadIdx.y+blockIdx.y*blockDim.y +vol->Ystart;
	int z = threadIdx.z+blockIdx.z*blockDim.z +vol->Zstart;
	//printf("%d %d\n ",y,z);
	if(x>=vol->Xend || y>=vol->Yend ||z>=vol->Zend) return;
	int i = (x-vol->Xstart)+(y-vol->Ystart)*vol->X+(z-vol->Zstart)*vol->X*vol->Y;
	w_data[i] = GAMMA*(2*v_data[i]-z_data[i]) + (1-GAMMA)*w_data[i];
}

__global__ void initial_zdata(float *vol_real,float *w_data,float *z_data,Volume *vol)
{
	int x = threadIdx.x+blockIdx.x*blockDim.x +vol->Xstart;
	int y = threadIdx.y+blockIdx.y*blockDim.y +vol->Ystart;
	int z = threadIdx.z+blockIdx.z*blockDim.z +vol->Zstart;
	//printf("%d %d\n ",y,z);
	if(x>=vol->Xend || y>=vol->Yend ||z>=vol->Zend) return;
	int i = (x-vol->Xstart)+(y-vol->Ystart)*vol->X+(z-vol->Zstart)*vol->X*vol->Y;
	z_data[i] = 2*vol_real[i] - w_data[i];
}

