#include "hip/hip_runtime.h"
#include "read_write_mrc.h"
#include "atom.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "file_read_write.cu"
#include "sirt.cu"
//#include "mace.cu"

#define FALSE 0
#define TRUE 1
#define checkCudaErrors( a ) do { \
	if (hipSuccess != (a)) { \
	fprintf(stderr, "Cuda runtime error in line %d of file %s \
	: %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
	exit(EXIT_FAILURE); \
	} \
	} while(0);
using namespace std;

double iStart,iElaps;

int ITER_NUM = 4;
float SIRT_STEP_LENGTH = 0.2;
float MACE_STEP_LENGTH = 0.5;
int file_num=2;
int NCC_FLAG = 0;

void check_GPU_mem()
{
	size_t avail;
	size_t total;
	hipMemGetInfo(&avail,&total);
	size_t used = total - avail;
	printf("Used:%zu /Total:%zu --%f   Rest:%zu\n",used,total,(float)used/total,avail);
}

double cpuSecond(){
	struct timeval tp;
	gettimeofday(&tp,NULL);
	//sce + msec
	return (double)tp.tv_sec +(double )tp.tv_usec*1e-6;
}

long long  vol_pixel_num, * PrjXYAngN;
int iLenx = 8,iLeny = 8,iLenz = 8; 

void copyOnCPU(float *tar,float *sou,long long l)
{
	for(long long k=0;k<l;k++)
	{
		tar[k] = sou[k];
	}
}


int main(int argc,char *argv[])
{
	iStart = cpuSecond();

/************************select device*****************************/
//	hipSetDevice(0);
//	check_GPU_mem();
/********************************************************************/


/**************************read parameter **********/
	char * out_addr;
	char ** in_addr;
	char ** txbr_addr;

	for(int i=0;i<argc;i++)
	{
		if(!strcmp("-lx",argv[i])) iLenx = atoi(argv[i+1]);
		if(!strcmp("-ly",argv[i])) iLeny = atoi(argv[i+1]);
		if(!strcmp("-lz",argv[i])) iLenz = atoi(argv[i+1]);
		if(!strcmp("-ncc",argv[i])) NCC_FLAG = atoi(argv[i+1]);
		if(!strcmp("-n",argv[i])) ITER_NUM = atoi(argv[i+1]);
		if(!strcmp("-s",argv[i])) SIRT_STEP_LENGTH = atof(argv[i+1]);
		if(!strcmp("-m",argv[i])) MACE_STEP_LENGTH = atof(argv[i+1]);
		if(!strcmp("-of",argv[i])) out_addr = argv[i+1];
		if(!strcmp("-fn",argv[i]))
		{
			file_num = atoi(argv[i+1]);
			in_addr = (char **)malloc(sizeof(char *)*file_num);
			txbr_addr = (char **)malloc(sizeof(char *)*file_num);
			for(int j=0;j<file_num;j++)
			{
				in_addr[j] = argv[i+2+j*2];
				txbr_addr[j] = argv[i+3+j*2];
			}
		}
	}
	
	printf("TOTAL iterate times:%d\n",ITER_NUM);
	printf("OUT file address:%s\n",out_addr);
	//for(int i=0;i<file_num;i++) printf("%s \n",in_addr[i]);
/*************************Malloc data space and Read head data************************/
	Volume *vol;
	Projection **prj = (Projection **)malloc(sizeof(Projection)*file_num);
	MrcHeader **in_head = (MrcHeader **)malloc(sizeof(MrcHeader)*file_num);
	MrcHeader *out_head;
	hipMallocManaged((void **)&vol,sizeof(Volume));
	hipMallocManaged((void **)&out_head,sizeof(MrcHeader));
	for(int i=0;i<file_num;i++)
	{
		hipMallocManaged((void **)&prj[i],sizeof(Projection));
		hipMallocManaged((void **)&in_head[i],sizeof(MrcHeader));
	}
	vol->Xstart = INF;
	vol->Xend = -INF;
	vol->Ystart = INF;
	vol->Yend = -INF;
	vol->Zstart = INF;
	vol->Zend = -INF;

	for(int i=0;i<file_num;i++)
		read_head_data(prj[i],in_head[i],in_addr[i]);
/******************************************************************************/

/********************read txbr file**********************************************/
	double **x_coef = (double **)malloc(sizeof(double *)*file_num);
	double **y_coef = (double **)malloc(sizeof(double *)*file_num);	

	for(int i=0;i<file_num;i++)
	{
		hipMallocManaged((void **)&x_coef[i],sizeof(double)*prj[i]->AngN*10);
		memset(x_coef[i], 0 , sizeof(double)*prj[i]->AngN*10);
		//printf("%d",sizeof(double)*prj->AngN*10);
		hipMallocManaged((void **)&y_coef[i],sizeof(double)*prj[i]->AngN*10);
		memset(y_coef[i], 0 , sizeof(double)*prj[i]->AngN*10);
	
		read_txbr_data(vol,x_coef[i],y_coef[i],txbr_addr[i]);	
	}
	vol->X = vol->Xend - vol->Xstart;
	vol->Y = vol->Yend - vol->Ystart;
	vol->Z = vol->Zend - vol->Zstart;
	printf("xs:%d xe:%d x:%d\n",vol->Xstart,vol->Xend,vol->X);
	printf("ys:%d ye:%d y:%d\n",vol->Ystart,vol->Yend,vol->Y);
	printf("zs:%d ze:%d z:%d\n",vol->Zstart,vol->Zend,vol->Z);
/*************************************************************/


/*************read all data*************************************************/
	float **prj_real = (float **)malloc(sizeof(float *)*file_num);
	vol_pixel_num = vol->X*(long long)vol->Y*vol->Z;
	printf("vol_pixel_num:%lld\n",vol_pixel_num);
	PrjXYAngN = (long long *)malloc(sizeof(long long)*file_num);
	long long PrjXYA =  -1;
	/*for input file*/
	for(int i=0;i<file_num;i++)
	{	
		PrjXYAngN[i] = prj[i]->X*(long long)prj[i]->Y*prj[i]->AngN;
		PrjXYA = max(PrjXYA,PrjXYAngN[i]);
	//	printf("%d PrjXYAngN:%lld\n",i,PrjXYAngN[i]);
		prj_real[i] = (float *)malloc(sizeof(float)*PrjXYAngN[i]);
		//checkCudaErrors(hipMallocManaged((void **)&prj_real[i],sizeof(float)*PrjXYAngN[i]));
		memset(prj_real[i], 0 , sizeof(float)*PrjXYAngN[i]);
		read_all_data(in_head[i],prj_real[i], in_addr[i]);
	}
/*******************************************************************/

/*****************************initial model*****************************/
	float **w_data = (float **)malloc(sizeof(float)*vol_pixel_num*file_num);
	float *z_data;
	float *v_data;
	for(int i=0;i<file_num;i++)
	{
		w_data[i] = (float *)malloc(sizeof(float)*vol_pixel_num);
	//	hipMallocManaged((void **)&w_data[i],sizeof(float)*vol_pixel_num);
	}	
	hipMallocManaged((void **)&z_data,sizeof(float)*vol_pixel_num);
	hipMallocManaged((void **)&v_data,sizeof(float)*vol_pixel_num);

	float *vol_real;
	hipMallocManaged((void **)&vol_real,sizeof(float)*vol_pixel_num);
	dim3 block(iLenx,iLeny,iLeny);
	dim3 grid_vol((vol->X+block.x-1)/block.x,(vol->Y+block.y-1)/block.y,(vol->Z+block.z-1)/block.z);
	
	float *dprj_real;
	checkCudaErrors(hipMallocManaged((void **)&dprj_real,sizeof(float)*PrjXYA));
	for(long long k=0;k<PrjXYAngN[0];k++) dprj_real[k] = prj_real[0][k];

	backProjOnGPU<<<grid_vol,block>>>(prj[0],vol,x_coef[0],y_coef[0],dprj_real,vol_real,1);
	hipDeviceSynchronize();
	for(int i=0;i<file_num;i++)	for(long long k=0;k<vol_pixel_num;k++) w_data[i][k] = vol_real[k];
/*************************************************************************/
	iElaps = cpuSecond()-iStart;
	printf("Host time elapsed:%lfsec\n",iElaps);	
/**********************************************IERTATION***************/
	float *dw_data;
	checkCudaErrors(hipMallocManaged((void **)&dw_data,sizeof(float)*vol_pixel_num));
	for(int i=0;i<ITER_NUM;i++)
	{
		for(int j=0;j<file_num;j++)
		{
			copyOnCPU(dw_data,w_data[j],vol_pixel_num);
			copyOnCPU(dprj_real,prj_real[j],PrjXYAngN[j]);
			
			//initial data
			initial_zdata<<<grid_vol,block>>>(vol_real,dw_data,z_data,vol);
			hipDeviceSynchronize();
			//sirt
			dim3 grid_prj((prj[j]->X+block.x-1)/block.x,(prj[j]->Y+block.y-1)/block.y,(prj[j]->AngN+block.z-1)/block.z);
			float *iter_prj_divisor,*iter_prj_dividend;
			checkCudaErrors(hipMallocManaged((void **)&iter_prj_divisor,sizeof(float)*PrjXYAngN[j]));
			checkCudaErrors(hipMallocManaged((void **)&iter_prj_dividend,sizeof(float)*PrjXYAngN[j]));	
			memset(iter_prj_divisor,0,sizeof(float)*PrjXYAngN[j]);
			memset(iter_prj_dividend,0,sizeof(float)*PrjXYAngN[j]);
			reProjOnGPU<<<grid_vol,block>>>(prj[j],vol,x_coef[j],y_coef[j],z_data,iter_prj_divisor,iter_prj_dividend);
			hipDeviceSynchronize();
			computePrjError<<<grid_prj,block>>>(prj[j],dprj_real,iter_prj_divisor,iter_prj_dividend);
			copyDataOnGPU<<<grid_vol,block>>>(v_data,z_data,vol);
			hipDeviceSynchronize();
			backProjOnGPU<<<grid_vol,block>>>(prj[j],vol,x_coef[j],y_coef[j],iter_prj_divisor,v_data,SIRT_STEP_LENGTH);
			hipDeviceSynchronize();
			//check_GPU_mem();
			
			//mace
			maceOnGPU<<<grid_vol,block>>>(v_data,dw_data,z_data,vol,MACE_STEP_LENGTH);	
			//copyDataOnGPU<<<grid_vol,block>>>(dw_data,v_data,vol);
			hipDeviceSynchronize();
			copyOnCPU(w_data[j],dw_data,vol_pixel_num);
		
			hipFree(iter_prj_divisor);
			hipFree(iter_prj_dividend);
		} 
		for(long long k=0;k<vol_pixel_num;k++)
		{
			vol_real[k] = 0;
			for(int j=0;j<file_num;j++) vol_real[k] += w_data[j][k];
			vol_real[k] /= file_num;
		}
		printf("Iteration %d finished..\n",i);
	}

/*****************************************************************************/
	iElaps = cpuSecond()-iStart;
	printf("Host time elapsed:%lfsec\n",iElaps);

/**********************  NCC  ************************************************/	
	if(NCC_FLAG)
	{
		float *iter_prj_divisor,*iter_prj_dividend;
		int j=0;
		checkCudaErrors(hipMallocManaged((void **)&iter_prj_divisor,sizeof(float)*PrjXYAngN[j]));
		checkCudaErrors(hipMallocManaged((void **)&iter_prj_dividend,sizeof(float)*PrjXYAngN[j]));	
		memset(iter_prj_divisor,0,sizeof(float)*PrjXYAngN[j]);
		memset(iter_prj_dividend,0,sizeof(float)*PrjXYAngN[j]);
		reProjOnGPU<<<grid_vol,block>>>(prj[j],vol,x_coef[j],y_coef[j],z_data,iter_prj_divisor,iter_prj_dividend);
		hipDeviceSynchronize();
		for(int k=0;k<PrjXYAngN[j];k++) if(iter_prj_dividend[k]!=0) iter_prj_divisor[k]/=iter_prj_dividend[k];

		double NCC_res[121];
		int prj_size = prj[j]->X*prj[j]->Y;
		for(int z=0;z<121;z++)
		{
			int st = prj_size*z;
			int ed = prj_size*(z+1);
			double mean1=0,mean2=0;
			for(int k=st;k<ed;k++)
			{
				mean1+=iter_prj_divisor[k];
				mean2+=prj_real[j][k];
			}
			mean1/=prj_size;
			mean2/=prj_size;
			double divisor=0;
			double dividend1=0,dividend2=0;
			for(int k=st;k<ed;k++)
			{
				divisor+=(iter_prj_divisor[k]-mean1)*(prj_real[j][k]-mean2);
				dividend1+=(iter_prj_divisor[k]-mean1)*(iter_prj_divisor[k]-mean1);
				dividend2+=(prj_real[j][k]-mean2)*(prj_real[j][k]-mean2);
			}
			NCC_res[z]=divisor/(sqrt(dividend1)*sqrt(dividend2));
		}
		printf("NCC result for %d iterations:\n",ITER_NUM);
		for(int z=0;z<121;z++) printf("%lf\n",NCC_res[z]);
	}
/*****************************************************************************/	


/***************************OUTPUT file******************************/
	mrc_init_head(out_head);
	set_head(out_head,vol);
	printf("OUT head:%d %d %d 0\n",out_head->nx,out_head->ny,out_head->nz);
	update_head(vol_real,out_head);
	write_data(out_addr,out_head,vol_real);

/*******************************************************************/

	hipDeviceReset();//重置CUDA设备释放程序占用的资源

	iElaps = cpuSecond()-iStart;
	printf("Host time elapsed:%lfsec\n",iElaps);
	return 0;
}


